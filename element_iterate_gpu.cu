#include "hip/hip_runtime.h"
#include"element_iterate_gpu.h"
#include<hip/hip_runtime.h>
#include<>
#include<sm_60_atomic_functions.h>
#include<thrust/device_vector.h>
#include<thrust/execution_policy.h>
#include<thrust/transform_scan.h>
#include<thrust/reverse.h>
#include<cmath>


constexpr int threads = 128;

__global__ void deltaS_iterate_kernel(double* deltaS_new, const double* deltaS_old, const double* velocity_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i>0 && i < size) deltaS_new[i] = deltaS_old[i] + dt * (velocity_old[i - 1] - velocity_old[i]);
}

__global__ void theta_iterate_kernel(double* theta_new, const double* theta_old, const double* omega_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) theta_new[i] = theta_old[i] + dt * omega_old[i];
}

__global__ void H_iterate_kernel(double* H_new, const double* H_old, const double* Delta_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		double H_temp = H_old[i], Delta_temp = Delta_old[i];
		double k1 = -H_temp * Delta_temp;
		double k2 = -(H_temp + dt / 2.0 * k1) * Delta_temp;
		double k3 = -(H_temp + dt / 2.0 * k2) * Delta_temp;
		double k4 = -(H_temp + dt * k3) * Delta_temp;
		H_new[i] = H_temp + dt / 6.0 * (k1 + 2.0 * k2 + 2.0 * k3 + k4);
	}
}

__global__ void K_iterate_kernel(double* K, const double* deltaS, const double* theta, const long long size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>0 && i < size - 1)
	{
		double dSj = deltaS[i];
		double dSj_1 = deltaS[i + 1];
		K[i] = theta[i + 1] * (-dSj) / (dSj_1 * (dSj_1 + dSj)) +
			theta[i] * (dSj - dSj_1) / (dSj * dSj_1) +
			theta[i - 1] * (dSj_1) / ((dSj + dSj_1) * dSj);
	}
}

template<typename Func>
__global__ void bodyforce_compute_kernel(double* GravityBase,const double* density, const double* H, const double* theta, const double g, const long long size, Func func)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) GravityBase[i] = H[i] * density[i] * g * func(theta[i]);
}

__global__ void omega_iterate_kernel(double* omega, const double* Omega, const double* deltaS,const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i], dSj_1 = deltaS[i + 1];
		omega[i-1] = Omega[i + 1] * (dSj + dSj_1) * (2.0 * dSj_1 - dSj) / 6.0 / dSj_1 +
			Omega[i] * (dSj + dSj_1) * (dSj + dSj_1) * (dSj + dSj_1) / 6.0 / dSj / dSj_1 +
			Omega[i - 1] * (dSj + dSj_1) * (2.0 * dSj - dSj_1) / 6.0 / dSj;
	}
}

__global__ void velocity_iterate_kernel(double* velocity,const double* Delta, const double* deltaS, const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i], dSj_1 = deltaS[i + 1];
		velocity[i-1] = Delta[i + 1] * (dSj + dSj_1) * (2.0 * dSj_1 - dSj) / 6.0 / dSj_1 +
			Delta[i] * (dSj + dSj_1) * (dSj + dSj_1) * (dSj + dSj_1) / 6.0 / dSj / dSj_1 +
			Delta[i - 1] * (dSj + dSj_1) * (2.0 * dSj - dSj_1) / 6.0 / dSj;
	}
}

__global__ void velocity_omega_iterate_kernel(double* velocity_or_omega, const double* Delta_or_Omega, const double* deltaS, const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i], dSj_1 = deltaS[i + 1];
		velocity_or_omega[i - 1] = Delta_or_Omega[i + 1] * (dSj + dSj_1) * (2.0 * dSj_1 - dSj) / 6.0 / dSj_1 +
			Delta_or_Omega[i] * (dSj + dSj_1) * (dSj + dSj_1) * (dSj + dSj_1) / 6.0 / dSj / dSj_1 +
			Delta_or_Omega[i - 1] * (dSj + dSj_1) * (2.0 * dSj - dSj_1) / 6.0 / dSj;
	}
}

template<typename Func>
__global__ void velocity_omega_aux_kernel(double* velocity_or_omega, const double* Delta_or_Omega, const double* deltaS, const long long size, Func func)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size-2)
	{
		double temp = (Delta_or_Omega[size - 1] + Delta_or_Omega[size - 2]) / 2.0 * deltaS[size - 1] + (Delta_or_Omega[i] + Delta_or_Omega[i + 1]) / 2.0 * deltaS[i + 1];
		velocity_or_omega[i] = func(velocity_or_omega[i], temp) / 2.0;
	}
}

__global__ void surface_force_sdirection_aux(double* b,const double * H,const double* Pup, const double* Pdown, const double * Tup, const double * Tdown,const double* deltaS,const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	long long n = size - 1;
	if (i > 0 && i < n)
	{
		double Hi = H[i], Hi_1 = H[i + 1], Pip = Pup[i], Pin = Pdown[i], Tip = Tup[i], Tin = Tdown[i],
			Pi_1p = Pup[i + 1], Pi_1n = Pdown[i + 1], Ti_1p = Tup[i + 1], Ti_1n = Tdown[i + 1],
			dSi_1 = deltaS[i + 1];
		b[2 * n + 1 - i] += -Tip + Tin - Hi / 2.0 / dSi_1 * (Pip + Pin - Pi_1p - Pi_1n);
	}
}


__global__ void surface_force_zdirection_aux(double* b, const double* H, const double* Pup, const double* Pdown, const double* Tup, const double* Tdown, const double* deltaS,const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	long long n = size - 1;
	if (i > 0 && i < n)
	{
		double Hi = H[i], Hi_1 = H[i + 1], Pip = Pup[i], Pin = Pdown[i], Tip = Tup[i], Tin = Tdown[i],
			Pi_1p = Pup[i + 1], Pi_1n = Pdown[i + 1], Ti_1p = Tup[i + 1], Ti_1n = Tdown[i + 1],
			dSi_1 = deltaS[i + 1];
		b[n - i] += -Pip + Pin - Hi / 2.0 / dSi_1 * (Tip + Tin - Ti_1p - Ti_1n) - (Hi - Hi_1) / dSi_1 * (Tip + Tin);
	}
}

extern "C"
{
	
	bool ElongateGpu(ElementGroup& egold, ElementGroup& egnew, ModelConf& model)
	{
		if (model.extrudepolicy.policy == ExtrudePolicy::Sparse)
		{
			if (!(model.extrudepolicy.iterating % model.extrudepolicy.SparseNum))
			{
				egold.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);
				egnew.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);
				model.extrudepolicy.iterating++;
				model.grid_num = egold.size;
				model.Standardize();
				return true;
			}
			model.extrudepolicy.iterating++;
			return false;
		}
		else if (model.extrudepolicy.policy == ExtrudePolicy::Dense)
		{

			egold.elongateGpu(model.extrudepolicy.Ds, model.H, model.velocity, model.extrudepolicy.DenseNum);
			egold.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);


			egnew.elongateGpu(model.extrudepolicy.Ds, model.H, model.velocity, model.extrudepolicy.DenseNum);
			egnew.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);

			model.grid_num = egold.size;
			model.Standardize();
			return true;
		}
	}

	void deltaS_iterate_gpu(ElementGroup& Egold,ElementGroup& Egnew,const double dt)
	{
		auto size = Egnew.size;
		//Egnew.deltaSGroup.SyncSize(HostToDevice()); Egold.deltaSGroup.send(); Egold.velocityGroup.send();
		deltaS_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.deltaSGroup.data(CVD),Egold.deltaSGroup.data(CVD), Egold.velocityGroup.data(CVD),Egold.size,dt);
	}

	void theta_iterate_gpu(ElementGroup& Egold, ElementGroup& Egnew, const double dt)
	{
		auto size = Egnew.size;
		//Egnew.thetaGroup.SyncSize(HostToDevice()); Egold.thetaGroup.send(); Egold.omegaGroup.send();
		theta_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.thetaGroup.data(CVD),Egold.thetaGroup.data(CVD),Egold.omegaGroup.data(CVD),Egold.size - 1,dt);
	}

	void H_iterate_gpu(ElementGroup& Egold, ElementGroup& Egnew,const double dt)
	{
		auto size = Egnew.size;
		//Egnew.HGroup.SyncSize(HostToDevice()); Egold.HGroup.send(); Egold.DeltaGroup.send();
		H_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.HGroup.data(CVD), Egold.HGroup.data(CVD), Egold.DeltaGroup.data(CVD),Egold.size, dt);
	}

	void deltaS_theta_H_synchronize(ElementGroup& Egnew)
	{
		hipDeviceSynchronize();
		//Egnew.deltaSGroup.fetch(); Egnew.thetaGroup.fetch(); Egnew.HGroup.fetch();
		Egnew.thetaGroup.back(CVD) = 0.0; Egnew.deltaSGroup(0) = Egnew.deltaSGroup(1);
	}

	void K_iterate_gpu(ElementGroup& Egnew)
	{
		//Egnew.KGroup.SyncSize(HostToDevice());
		auto size = Egnew.size;
		//compute the outside point 
		double dSn_1 = Egnew.deltaSGroup(1);
		double dSn_2 = Egnew.deltaSGroup(2);
		Egnew.KGroup(0) = Egnew.thetaGroup(2) * dSn_1 / (dSn_2 * (dSn_2 + dSn_1)) +
			Egnew.thetaGroup(1) * (-dSn_1 - dSn_2) / (dSn_1 * dSn_2) +
			Egnew.thetaGroup(0) * (dSn_2 + 2.0 * dSn_1) / ((dSn_1 + dSn_2) * dSn_1);

		//compute the inner point
		double dS0 = Egnew.deltaSGroup(size - 1);
		double dS1 = Egnew.deltaSGroup(size - 2);
		Egnew.KGroup(size - 1) = Egnew.thetaGroup(size - 1) * (-2.0 * dS0 - dS1) / ((dS0 + dS1) * dS0) +
			Egnew.thetaGroup(size - 2) * (dS0 + dS1) / (dS0 * dS1) +
			Egnew.thetaGroup(size - 3) * (-dS0) / ((dS1 + dS0) * dS1);

		K_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.KGroup.data(CVD), Egnew.deltaSGroup.data(CVD), Egnew.thetaGroup.data(CVD),size);
	}

	void density_iterate_gpu(ElementGroup& Egnew,ModelConf& model)
	{
		//Egnew.densityGroup.send();
	}

	void bodyforce_compute_gpu(ElementGroup& Egnew)
	{
		auto size = Egnew.size;
		//Egnew.GravityGroup.SyncSize(HostToDevice()); Egnew.GravityGroupCos.SyncSize(HostToDevice()); Egnew.GravityGroupSin.SyncSize(HostToDevice());
		bodyforce_compute_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.GravityGroup.data(CVD), Egnew.densityGroup.data(CVD), Egnew.HGroup.data(CVD), Egnew.thetaGroup.data(CVD), Egnew.g, Egnew.size, []__device__(double i) { return 1.0; });
		bodyforce_compute_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.GravityGroupCos.data(CVD), Egnew.densityGroup.data(CVD), Egnew.HGroup.data(CVD), Egnew.thetaGroup.data(CVD), Egnew.g, Egnew.size, [=] __device__(double i) { return cos(i); });
		bodyforce_compute_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.GravityGroupSin.data(CVD), Egnew.densityGroup.data(CVD), Egnew.HGroup.data(CVD), Egnew.thetaGroup.data(CVD), Egnew.g, Egnew.size, [=] __device__(double i) { return sin(i); });
	}

	void K_density_bodyforce_synchronize(ElementGroup& Egnew)
	{
		hipDeviceSynchronize();
		//Egnew.KGroup.trans(Egnew.KGroup.begin(CVD) + 1, Egnew.KGroup.end(CVD) - 1, Egnew.KGroup.begin() + 1); Egnew.GravityGroup.fetch(); Egnew.GravityGroupCos.fetch(); Egnew.GravityGroupSin.fetch();
	}

	void surface_force_iterate_gpu(ElementGroup& Egnew, ModelConf& model, int iterating)
	{

	}

	void Omega_Delta_iterate_gpu(ElementGroup& Egnew, ModelConf& model, SolverInterface* SolverHandle, bool ResetMatrix)
	{
		//the number of length element
		long long n = Egnew.size - 1;
		static CuVector<double> vals;  static CuVector<double> b; static CuVector<int> rowPtr; static CuVector<int> colInd;

		switch (model.boundaryCondition)
		{
		case BoundaryCondition::ClampedFree:
			if (ResetMatrix) ClampedFree(Egnew, vals, rowPtr, colInd); else ClampedFreeGpu(Egnew, vals, rowPtr, colInd); break;
		case BoundaryCondition::ClampedBoth:
			//ClampedBoth(Egnew, vals, rowPtr, colInd); 
			break;
		default:
			break;
		}
		

		if (ResetMatrix)
		{
			SolverHandle->Reset();
			SolverHandle->Initialize(vals.data(), rowPtr.data(), colInd.data(),vals.size(),rowPtr.size());
		}
		else
		{
			reinterpret_cast<CusolverRfHandle*>(SolverHandle)->ResetAGpu(vals.data(CVD), rowPtr.data(CVD), colInd.data(CVD),vals.size(CVD),rowPtr.size(CVD));
		}
		
		thrust::for_each(SolverHandle->X.begin(CVD), SolverHandle->X.end(CVD), []__device__(auto & it) { it = 0.0; });

		switch (model.forceCondition)
		{
		case ForceCondition::BodyForceOnly:
			BodyForceGpu(Egnew, SolverHandle->X); break;
		case ForceCondition::SurfaceAndBodyForce:
			SurfaceForceGpu(Egnew, SolverHandle->X); BodyForceGpu(Egnew, SolverHandle->X); break;
		case ForceCondition::SurfaceForceOnly:
			SurfaceForceGpu(Egnew, SolverHandle->X); break;
		default:
			break;
		}

		SolverHandle->solve();
	}

	void omega_velocity_iterate_gpu(ElementGroup& Egnew, ModelConf& model,SolverInterface* handle)
	{
		auto size = Egnew.size;
		//Egnew.omegaGroup.SyncSize(HostToDevice());
		//Egnew.velocityGroup.SyncSize(HostToDevice());
		
		thrust::reverse(thrust::device, handle->X.begin(CVD), handle->X.end(CVD));
		thrust::transform(thrust::device, handle->X.begin(CVD), handle->X.begin(CVD) + size, Egnew.HGroup.begin(CVD), handle->X.begin(CVD), []__device__(auto & it1, auto & it2) { return it1 / it2; });
		thrust::transform(thrust::device, handle->X.begin(CVD) + size, handle->X.end(CVD), Egnew.HGroup.begin(CVD), handle->X.begin(CVD) + size, []__device__(auto & it1, auto & it2) { return it1 / it2 / it2 / it2; });
		//thrust::copy(handle->X.begin(CVD), handle->X.begin(CVD) + size, Egnew.DeltaGroup.data());
		//thrust::copy(handle->X.begin(CVD) + size, handle->X.end(CVD), Egnew.OmegaGroup.data());
		
		Egnew.omegaGroup(size - 1) = 0.0;
		Egnew.omegaGroup(size - 2) = -(Egnew.OmegaGroup(size - 1) + Egnew.OmegaGroup(size - 2)) * Egnew.deltaSGroup(size - 1) / 2.0;
		Egnew.velocityGroup(size - 1) = 0.0;
		Egnew.velocityGroup(size - 2) = (Egnew.DeltaGroup(size - 1) + Egnew.DeltaGroup(size - 2)) * Egnew.deltaSGroup(size - 1) / 2.0;
		velocity_omega_iterate_kernel << <(size + threads - 1) / threads, threads >> > (Egnew.omegaGroup.data(CVD),handle->X.data(CVD) + size,Egnew.deltaSGroup.data(CVD),size);
		velocity_omega_iterate_kernel << <(size + threads - 1) / threads, threads >> > (Egnew.velocityGroup.data(CVD),handle->X.data(CVD), Egnew.deltaSGroup.data(CVD),size);
		hipDeviceSynchronize();
		//0~size-3
		
		thrust::transform_inclusive_scan(Egnew.omegaGroup.rbegin(CVD)+2, Egnew.omegaGroup.rend(CVD), Egnew.omegaGroup.rbegin(CVD)+2, thrust::negate<double>(), thrust::plus<double>());
		thrust::inclusive_scan(Egnew.velocityGroup.rbegin(CVD) + 2, Egnew.velocityGroup.rend(CVD), Egnew.velocityGroup.rbegin(CVD) + 2, thrust::plus<double>());
		velocity_omega_aux_kernel << < (size + threads - 1) / threads, threads >> > (Egnew.velocityGroup.data(CVD), handle->X.data(CVD), Egnew.deltaSGroup.data(CVD), size, thrust::plus<double>());
		velocity_omega_aux_kernel << < (size + threads - 1) / threads, threads >> > (Egnew.omegaGroup.data(CVD), handle->X.data(CVD) + size, Egnew.deltaSGroup.data(CVD), size, thrust::minus<double>());
		hipDeviceSynchronize();
		
		double C;
		//if(model.omegaStandard.first < size - 2)
		C = model.omegaStandard.second - Egnew.omegaGroup(model.omegaStandard.first);
		//else 
		//	C = model.omegaStandard.second - Egnew.omegaGroup[model.omegaStandard.first];
		thrust::for_each(Egnew.omegaGroup.begin(CVD), Egnew.omegaGroup.end(CVD), [=]__device__(double& it) {it += C; });
		//Egnew.omegaGroup[size - 1] += C; Egnew.omegaGroup[size - 2] += C;
		//if(model.velocityStandard.first < size - 2)
		C = model.velocityStandard.second - Egnew.velocityGroup(model.velocityStandard.first);
		//else
		//	C = model.velocityStandard.second - Egnew.velocityGroup[model.velocityStandard.first];
		thrust::for_each(Egnew.velocityGroup.begin(CVD), Egnew.velocityGroup.end(CVD), [=]__device__(double& it) {it += C; });
		//Egnew.velocityGroup[size - 1] += C; Egnew.velocityGroup[size - 2] += C;
		
		//Egnew.velocityGroup.trans(Egnew.velocityGroup.begin(CVD), Egnew.velocityGroup.end(CVD) - 2, Egnew.velocityGroup.begin());
		//Egnew.omegaGroup.trans(Egnew.omegaGroup.begin(CVD), Egnew.omegaGroup.end(CVD) - 2, Egnew.omegaGroup.begin());
		
	}

	void ClampedFreeGpu(ElementGroup& eg, CuVector<double>& vals, CuVector<int>& rowPtr, CuVector<int>& colInd)
	{
	
		long long n = eg.size - 1;
		vals.erase(CVD); colInd.erase(CVD); rowPtr.erase(CVD);
		vals.resize(10 * n - 2); colInd.resize(10 * n - 2); rowPtr.resize(2 * n + 3);
		
		
		colInd.push_back(2 * n - 1);
		colInd.push_back(2 * n);
		colInd.push_back(2 * n + 1);
		for (long long i = 1; i < n; i++)
		{
			colInd.push_back(i - 1);
			colInd.push_back(i);
			colInd.push_back(i + 1);

			colInd.push_back(i + n + 1);

		}
		colInd.push_back(n - 2);
		colInd.push_back(n - 1);
		colInd.push_back(n);

		//colInd.push_back(n + 1);
		colInd.push_back(n);
		for (long long i = 1; i < n; i++)
		{
			colInd.push_back(i - 1);
			colInd.push_back(i);
			colInd.push_back(i + 1);
			//

			colInd.push_back(i + n);
			colInd.push_back(i + n + 1);
			colInd.push_back(i + n + 2);
		}
		colInd.push_back(2 * n + 1);


		long long sum = 0;
		rowPtr.push_back(0);
		sum += 3;
		rowPtr.push_back(sum);
		for (long long i = 0; i < n - 1; i++)
		{
			sum += 4;
			rowPtr.push_back(sum);
		}
		sum += 3;
		rowPtr.push_back(sum);

		sum += 1;
		rowPtr.push_back(sum);
		for (long long i = 0; i < n - 1; i++)
		{
			//
			sum += 6;
			rowPtr.push_back(sum);
		}
		sum += 1;
		rowPtr.push_back(sum);
		

		double dSn_2 = eg.deltaSGroup[2], dSn_1 = eg.deltaSGroup[1];
		vals.push_back(dSn_1 / dSn_2 / (dSn_1 + dSn_2) / eg.HGroup[2]);
		vals.push_back((-dSn_1 - dSn_2) / dSn_1 / dSn_2 / eg.HGroup[1]);
		vals.push_back((dSn_2 + 2.0 * dSn_1) / dSn_1 / (dSn_1 + dSn_2) / eg.HGroup[0]);
		for (long long i = n - 1; i > 0; i--)
		{
			double dSi = eg.deltaSGroup[i], dSi_1 = eg.deltaSGroup[i + 1], Ki = eg.KGroup[i], miu = eg.viscosity;
			vals.push_back(miu / 3.0 * 2.0 / dSi_1 / (dSi + dSi_1));
			vals.push_back(-miu / 3.0 * 2.0 / dSi / dSi_1 + 5.0 * miu / 6.0 * Ki * Ki);
			vals.push_back(miu / 3.0 * 2.0 / dSi / (dSi + dSi_1));
			vals.push_back(4.0 * miu * Ki);
		}


		vals.push_back(dSn_1 / dSn_2 / (dSn_1 + dSn_2) / eg.HGroup[2] / eg.HGroup[2] / eg.HGroup[2]);
		vals.push_back((-dSn_1 - dSn_2) / dSn_1 / dSn_2 / eg.HGroup[1] / eg.HGroup[1] / eg.HGroup[1]);
		vals.push_back((dSn_2 + 2.0 * dSn_1) / dSn_1 / (dSn_1 + dSn_2) / eg.HGroup[0] / eg.HGroup[0] / eg.HGroup[0]);

		vals.push_back(1.0);
		for (long long i = n - 1; i > 0; i--)
		{
			double dSi = eg.deltaSGroup[i], dSi_1 = eg.deltaSGroup[i + 1], Ki = eg.KGroup[i], Ki_1 = eg.KGroup[i + 1], Kip1 = eg.KGroup[i - 1], miu = eg.viscosity;
			vals.push_back(-miu / 2.0 * Ki * dSi / dSi_1 / (dSi_1 + dSi));
			vals.push_back(miu / 2.0 * Ki * (dSi - dSi_1) / dSi / dSi_1 + 5.0 * miu / 6.0 / dSi / dSi_1 / (dSi_1 + dSi) * (Kip1 * dSi_1 * dSi_1 - Ki_1 * dSi * dSi + Ki * (dSi * dSi - dSi_1 * dSi_1)));
			vals.push_back(miu / 2.0 * Ki * dSi_1 / dSi / (dSi_1 + dSi));

			vals.push_back(-4.0 * miu * dSi / dSi_1 / (dSi + dSi_1));
			vals.push_back(4.0 * miu * (dSi - dSi_1) / dSi / dSi_1);
			vals.push_back(4.0 * miu * dSi_1 / dSi / (dSi_1 + dSi));
		}
		vals.push_back(1.0);

	}

	void BodyForceGpu(ElementGroup& eg, CuVector<double>& b)
	{
		long long n = eg.size - 1;
		thrust::transform(eg.GravityGroupCos.rbegin(CVD) + 1, eg.GravityGroupCos.rend(CVD) - 1, b.begin(CVD) + 1, b.begin(CVD) + 1,thrust::plus<double>());
		thrust::transform(eg.GravityGroupSin.rbegin(CVD) + 1, eg.GravityGroupSin.rend(CVD) - 1, b.begin(CVD) + 2 + n, b.begin(CVD) + 2 + n,thrust::plus<double>());
		//b(0) = 0.0; b(n) = 0.0; b(n + 1) = 0.0; b(2 * n + 1) = 0.0;
	}

	void SurfaceForceGpu(ElementGroup& eg, CuVector<double>& b)
	{
		auto size = eg.size;
		surface_force_sdirection_aux << < (size + threads - 1) / threads, threads >> > (b.data(CVD), eg.HGroup.data(CVD), eg.PupGroup.data(CVD), eg.PdownGroup.data(CVD), eg.TupGroup.data(CVD), eg.TdownGroup.data(CVD), eg.deltaSGroup.data(CVD), eg.size);
		surface_force_zdirection_aux << < (size + threads - 1) / threads, threads >> > (b.data(CVD), eg.HGroup.data(CVD), eg.PupGroup.data(CVD), eg.PdownGroup.data(CVD), eg.TupGroup.data(CVD), eg.TdownGroup.data(CVD), eg.deltaSGroup.data(CVD), eg.size);
		hipDeviceSynchronize();
	}


}//end of extern "C"