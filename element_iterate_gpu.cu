#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<sm_60_atomic_functions.h>
#include"element_iterate_gpu.h"
#include<cmath>


__global__ void deltaS_iterate_kernel(double* deltaS_new, const double* deltaS_old, const double* velocity_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)return;
	if(i>0) deltaS_new[i] = deltaS_old[i] + dt * (velocity_old[i - 1] - velocity_old[i]);
}

__global__ void theta_iterate_kernel(double* theta_new, const double* theta_old, const double* omega_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)return;
	theta_new[i] = theta_old[i] + dt * omega_old[i];
}

__global__ void H_iterate_kernel(double* H_new, const double* H_old, const double* Delta_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)return;
	double k1 = -H_old[i] * Delta_old[i];
	double k2 = -(H_old[i] + dt / 2.0 * k1) * Delta_old[i];
	double k3 = -(H_old[i] + dt / 2.0 * k2) * Delta_old[i];
	double k4 = -(H_old[i] + dt * k3) * Delta_old[i];
	H_new[i] = H_old[i] + dt / 6.0 * (k1 + 2.0 * k2 + 2.0 * k3 + k4);
}

__global__ void K_iterate_kernel(double* K, const double* deltaS, const double* theta, const long long size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i];
		double dSj_1 = deltaS[i + 1];
		K[i] = theta[i + 1] * (-dSj) / (dSj_1 * (dSj_1 + dSj)) +
			theta[i] * (dSj - dSj_1) / (dSj * dSj_1) +
			theta[i - 1] * (dSj_1) / ((dSj + dSj_1) * dSj);
	}
}

template<typename Func>
__global__ void bodyforce_compute_kernel(double* GravityBase,const double* density, const double* H, const double* theta, const double g, const long long size, Func func)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size)return;
	GravityBase[i] = H[i] * density[i] * g * func(theta[i]);
}

__global__ void omega_iterate_kernel(double* omega, const double* Omega, const double* deltaS, const long long size)
{

}

__global__ void velocity_iterate_kernel(double* velocity, const double* Delta, const double* deltaS, const long long size)
{

}

extern "C"
{
	void deltaS_iterate_gpu(double* deltaS_new, const double* deltaS_old, const double* velocity_old, const long long size, const double dt)
	{
		//hipMemPrefetchAsync()
		deltaS_iterate_kernel << <(size + 63) / 64, 64 >> > (deltaS_new,deltaS_old, velocity_old,size,dt);
	}

	void theta_iterate_gpu(double* theta_new, const double* theta_old, const double* omega_old, const long long size, const double dt)
	{
		theta_iterate_kernel << <(size + 63) / 64, 64 >> > (theta_new,theta_old,omega_old,size - 1,dt);
	}

	void H_iterate_gpu(double* H_new, const double* H_old, const double* Delta_old, const long long size, const double dt)
	{
		H_iterate_kernel << <(size + 63) / 64, 64 >> > (H_new, H_old, Delta_old, size, dt);
	}

	void K_iterate_gpu(double* K, const double* deltaS, const double* theta, const long long size)
	{
		//compute the outside point 
		double dSn_1 = deltaS[1];
		double dSn_2 = deltaS[2];
		K[0] = theta[2] * dSn_1 / (dSn_2 * (dSn_2 + dSn_1)) +
			theta[1] * (-dSn_1 - dSn_2) / (dSn_1 * dSn_2) +
			theta[0] * (dSn_2 + 2.0 * dSn_1) / ((dSn_1 + dSn_2) * dSn_1);

		//compute the inner point
		double dS0 = deltaS[size - 1];
		double dS1 = deltaS[size - 2];
		K[size - 1] = theta[size - 1] * (-2.0 * dS0 - dS1) / ((dS0 + dS1) * dS0) +
			theta[size - 2] * (dS0 + dS1) / (dS0 * dS1) +
			theta[size - 3] * (-dS0) / ((dS1 + dS0) * dS1);

		K_iterate_kernel << <(size + 63) / 64, 64 >> > (K, deltaS, theta, size);
	}

	void bodyforce_compute_gpu(double* Gravity, double* GravityCos, double* GravitySin, const double* density, const double* H, const double* theta, const double g, const long long size)
	{
		bodyforce_compute_kernel << <(size + 63) / 64, 64 >> > (Gravity, density, H, theta, g, size, []__device__(double i) { return 1.0; });
		bodyforce_compute_kernel << <(size + 63) / 64, 64 >> > (GravityCos, density, H, theta, g, size, [=] __device__(double i) { return cos(i); });
		bodyforce_compute_kernel << <(size + 63) / 64, 64 >> > (GravitySin, density, H, theta, g, size, [=] __device__(double i) { return sin(i); });
	}

	void omega_iterate_gpu(double* omega, const double* Omega, const double* deltaS, const long long size)
	{
		omega[size - 1] = 0.0;
		omega[size - 2] = -(Omega[size - 1] + Omega[size - 2]) * deltaS[size - 1] / 2.0;
		omega_iterate_kernel << <(size + 63) / 64, 64 >> > (omega, Omega, deltaS, size);
	}

	void velocity_iterate_gpu(double* velocity, const double* Delta, const double* deltaS, const long long size)
	{
		velocity[size - 1] = 0.0;
		velocity[size - 2] = (Delta[size - 1] + Delta[size - 2]) * deltaS[size - 1] / 2.0;
		velocity_iterate_kernel << <(size + 63) / 64, 64 >> > (velocity, Delta, deltaS, size);
	}
}