#include "hip/hip_runtime.h"
#include"element_iterate_gpu.h"
#include<hip/hip_runtime.h>
#include<>
#include<sm_60_atomic_functions.h>
#include<thrust/device_vector.h>
#include<thrust/execution_policy.h>
#include<thrust/transform_scan.h>
#include<thrust/reverse.h>
#include<cmath>


constexpr int threads = 128;

__global__ void deltaS_iterate_kernel(double* deltaS_new, const double* deltaS_old, const double* velocity_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i>0 && i < size) deltaS_new[i] = deltaS_old[i] + dt * (velocity_old[i - 1] - velocity_old[i]);
}

__global__ void theta_iterate_kernel(double* theta_new, const double* theta_old, const double* omega_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) theta_new[i] = theta_old[i] + dt * omega_old[i];
}

__global__ void H_iterate_kernel(double* H_new, const double* H_old, const double* Delta_old, const long long size, const double dt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		double H_temp = H_old[i], Delta_temp = Delta_old[i];
		double k1 = -H_temp * Delta_temp;
		double k2 = -(H_temp + dt / 2.0 * k1) * Delta_temp;
		double k3 = -(H_temp + dt / 2.0 * k2) * Delta_temp;
		double k4 = -(H_temp + dt * k3) * Delta_temp;
		H_new[i] = H_temp + dt / 6.0 * (k1 + 2.0 * k2 + 2.0 * k3 + k4);
	}
}

__global__ void K_iterate_kernel(double* K, const double* deltaS, const double* theta, const long long size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>0 && i < size - 1)
	{
		double dSj = deltaS[i];
		double dSj_1 = deltaS[i + 1];
		K[i] = theta[i + 1] * (-dSj) / (dSj_1 * (dSj_1 + dSj)) +
			theta[i] * (dSj - dSj_1) / (dSj * dSj_1) +
			theta[i - 1] * (dSj_1) / ((dSj + dSj_1) * dSj);
	}
}

template<typename Func>
__global__ void bodyforce_compute_kernel(double* GravityBase,const double* density, const double* H, const double* theta, const double g, const long long size, Func func)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) GravityBase[i] = H[i] * density[i] * g * func(theta[i]);
}

__global__ void omega_iterate_kernel(double* omega, const double* Omega, const double* deltaS,const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i], dSj_1 = deltaS[i + 1];
		omega[i-1] = Omega[i + 1] * (dSj + dSj_1) * (2.0 * dSj_1 - dSj) / 6.0 / dSj_1 +
			Omega[i] * (dSj + dSj_1) * (dSj + dSj_1) * (dSj + dSj_1) / 6.0 / dSj / dSj_1 +
			Omega[i - 1] * (dSj + dSj_1) * (2.0 * dSj - dSj_1) / 6.0 / dSj;
	}
}

__global__ void velocity_iterate_kernel(double* velocity,const double* Delta, const double* deltaS, const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i], dSj_1 = deltaS[i + 1];
		velocity[i-1] = Delta[i + 1] * (dSj + dSj_1) * (2.0 * dSj_1 - dSj) / 6.0 / dSj_1 +
			Delta[i] * (dSj + dSj_1) * (dSj + dSj_1) * (dSj + dSj_1) / 6.0 / dSj / dSj_1 +
			Delta[i - 1] * (dSj + dSj_1) * (2.0 * dSj - dSj_1) / 6.0 / dSj;
	}
}

__global__ void velocity_omega_iterate_kernel(double* velocity_or_omega, const double* Delta_or_Omega, const double* deltaS, const long long size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > 0 && i < size - 1)
	{
		double dSj = deltaS[i], dSj_1 = deltaS[i + 1];
		velocity_or_omega[i - 1] = Delta_or_Omega[i + 1] * (dSj + dSj_1) * (2.0 * dSj_1 - dSj) / 6.0 / dSj_1 +
			Delta_or_Omega[i] * (dSj + dSj_1) * (dSj + dSj_1) * (dSj + dSj_1) / 6.0 / dSj / dSj_1 +
			Delta_or_Omega[i - 1] * (dSj + dSj_1) * (2.0 * dSj - dSj_1) / 6.0 / dSj;
	}
}

template<typename Func>
__global__ void velocity_omega_aux_kernel(double* velocity_or_omega, const double* Delta_or_Omega, const double* deltaS, const long long size, Func func)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size-2)
	{
		double temp = (Delta_or_Omega[size - 1] + Delta_or_Omega[size - 2]) / 2.0 * deltaS[size - 1] + (Delta_or_Omega[i] + Delta_or_Omega[i + 1]) / 2.0 * deltaS[i + 1];
		velocity_or_omega[i] = func(velocity_or_omega[i], temp) / 2.0;
	}
}

extern "C"
{
	bool ElongateGpu(ElementGroup& egold, ElementGroup& egnew, ModelConf& model)
	{
		if (model.extrudepolicy.policy == ExtrudePolicy::Sparse)
		{
			if (!(model.extrudepolicy.iterating % model.extrudepolicy.SparseNum))
			{
				egold.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);
				egnew.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);
				model.extrudepolicy.iterating++;
				model.grid_num = egold.size;
				model.Standardize();
				return true;
			}
			model.extrudepolicy.iterating++;
			return false;
		}
		else if (model.extrudepolicy.policy == ExtrudePolicy::Dense)
		{

			egold.elongateGpu(model.extrudepolicy.Ds, model.H, model.velocity, model.extrudepolicy.DenseNum);
			egold.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);


			egnew.elongateGpu(model.extrudepolicy.Ds, model.H, model.velocity, model.extrudepolicy.DenseNum);
			egnew.elongateGpu(model.extrudepolicy.DsEnd, model.H, model.velocity);

			model.grid_num = egold.size;
			model.Standardize();
			return true;
		}
	}

	void deltaS_iterate_gpu(ElementGroup& Egold,ElementGroup& Egnew,const double dt)
	{
		auto size = Egnew.size;
		//Egnew.deltaSGroup.SyncSize(HostToDevice()); Egold.deltaSGroup.send(); Egold.velocityGroup.send();
		deltaS_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.deltaSGroup.data(CVD),Egold.deltaSGroup.data(CVD), Egold.velocityGroup.data(CVD),Egold.size,dt);
	}

	void theta_iterate_gpu(ElementGroup& Egold, ElementGroup& Egnew, const double dt)
	{
		auto size = Egnew.size;
		//Egnew.thetaGroup.SyncSize(HostToDevice()); Egold.thetaGroup.send(); Egold.omegaGroup.send();
		theta_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.thetaGroup.data(CVD),Egold.thetaGroup.data(CVD),Egold.omegaGroup.data(CVD),Egold.size - 1,dt);
	}

	void H_iterate_gpu(ElementGroup& Egold, ElementGroup& Egnew,const double dt)
	{
		auto size = Egnew.size;
		//Egnew.HGroup.SyncSize(HostToDevice()); Egold.HGroup.send(); Egold.DeltaGroup.send();
		H_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.HGroup.data(CVD), Egold.HGroup.data(CVD), Egold.DeltaGroup.data(CVD),Egold.size, dt);
	}

	void deltaS_theta_H_synchronize(ElementGroup& Egnew)
	{
		hipDeviceSynchronize();
		//Egnew.deltaSGroup.fetch(); Egnew.thetaGroup.fetch(); Egnew.HGroup.fetch();
		Egnew.thetaGroup.back(CVD) = 0.0; Egnew.deltaSGroup.Dvec[0] = Egnew.deltaSGroup.Dvec[1];
	}

	void K_iterate_gpu(ElementGroup& Egnew)
	{
		//Egnew.KGroup.SyncSize(HostToDevice());
		auto size = Egnew.size;
		//compute the outside point 
		double dSn_1 = Egnew.deltaSGroup.Dvec[1];
		double dSn_2 = Egnew.deltaSGroup.Dvec[2];
		Egnew.KGroup.Dvec[0] = Egnew.thetaGroup.Dvec[2] * dSn_1 / (dSn_2 * (dSn_2 + dSn_1)) +
			Egnew.thetaGroup.Dvec[1] * (-dSn_1 - dSn_2) / (dSn_1 * dSn_2) +
			Egnew.thetaGroup.Dvec[0] * (dSn_2 + 2.0 * dSn_1) / ((dSn_1 + dSn_2) * dSn_1);

		//compute the inner point
		double dS0 = Egnew.deltaSGroup.Dvec[size - 1];
		double dS1 = Egnew.deltaSGroup.Dvec[size - 2];
		Egnew.KGroup.Dvec[size - 1] = Egnew.thetaGroup.Dvec[size - 1] * (-2.0 * dS0 - dS1) / ((dS0 + dS1) * dS0) +
			Egnew.thetaGroup.Dvec[size - 2] * (dS0 + dS1) / (dS0 * dS1) +
			Egnew.thetaGroup.Dvec[size - 3] * (-dS0) / ((dS1 + dS0) * dS1);

		K_iterate_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.KGroup.data(CVD), Egnew.deltaSGroup.data(CVD), Egnew.thetaGroup.data(CVD),size);
	}

	void density_iterate_gpu(ElementGroup& Egnew,ModelConf& model)
	{
		//Egnew.densityGroup.send();
	}

	void bodyforce_compute_gpu(ElementGroup& Egnew)
	{
		auto size = Egnew.size;
		//Egnew.GravityGroup.SyncSize(HostToDevice()); Egnew.GravityGroupCos.SyncSize(HostToDevice()); Egnew.GravityGroupSin.SyncSize(HostToDevice());
		bodyforce_compute_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.GravityGroup.data(CVD), Egnew.densityGroup.data(CVD), Egnew.HGroup.data(CVD), Egnew.thetaGroup.data(CVD), Egnew.g, Egnew.size, []__device__(double i) { return 1.0; });
		bodyforce_compute_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.GravityGroupCos.data(CVD), Egnew.densityGroup.data(CVD), Egnew.HGroup.data(CVD), Egnew.thetaGroup.data(CVD), Egnew.g, Egnew.size, [=] __device__(double i) { return cos(i); });
		bodyforce_compute_kernel << <(size +  threads - 1) / threads, threads >> > (Egnew.GravityGroupSin.data(CVD), Egnew.densityGroup.data(CVD), Egnew.HGroup.data(CVD), Egnew.thetaGroup.data(CVD), Egnew.g, Egnew.size, [=] __device__(double i) { return sin(i); });
	}

	void K_density_bodyforce_synchronize(ElementGroup& Egnew)
	{
		hipDeviceSynchronize();
		//Egnew.KGroup.trans(Egnew.KGroup.begin(CVD) + 1, Egnew.KGroup.end(CVD) - 1, Egnew.KGroup.begin() + 1); Egnew.GravityGroup.fetch(); Egnew.GravityGroupCos.fetch(); Egnew.GravityGroupSin.fetch();
	}

	void surface_force_iterate_gpu(ElementGroup& Egnew, ModelConf& model, int iterating)
	{

	}

	void Omega_Delta_iterate_gpu(ElementGroup& Egnew, ModelConf& model, SolverInterface* SolverHandle, bool ResetMatrix)
	{
		//the number of length element
		long long n = Egnew.size - 1;
		static std::vector<double> vals;  static std::vector<double> b; static std::vector<int> rowPtr; static std::vector<int> colInd;

		switch (model.boundaryCondition)
		{
		case BoundaryCondition::ClampedFree:
			ClampedFree(Egnew, vals, rowPtr, colInd); break;
		case BoundaryCondition::ClampedBoth:
			ClampedBoth(Egnew, vals, rowPtr, colInd); break;
		default:
			break;
		}
		switch (model.forceCondition)
		{
		case ForceCondition::BodyForceOnly:
			BodyForceOnly(Egnew, b); break;
		case ForceCondition::SurfaceAndBodyForce:
			SurfaceAndBodyForce(Egnew, b); break;
		case ForceCondition::SurfaceForceOnly:
			SurfaceForceOnly(Egnew, b); break;
		default:
			break;
		}


		if (ResetMatrix)
		{
			SolverHandle->Reset();
			SolverHandle->Initialize(vals, rowPtr, colInd);
		}
		else
		{
			SolverHandle->ResetA(vals, rowPtr, colInd);
		}
		SolverHandle->loadB(b);
		SolverHandle->solve();
	}

	void omega_velocity_iterate_gpu(ElementGroup& Egnew, ModelConf& model,SolverInterface* handle)
	{
		auto size = Egnew.size;
		//Egnew.omegaGroup.SyncSize(HostToDevice());
		//Egnew.velocityGroup.SyncSize(HostToDevice());
		
		thrust::reverse(thrust::device, handle->X.begin(CVD), handle->X.end(CVD));
		thrust::transform(thrust::device, handle->X.begin(CVD), handle->X.begin(CVD) + size, Egnew.HGroup.begin(CVD), handle->X.begin(CVD), []__device__(auto & it1, auto & it2) { return it1 / it2; });
		thrust::transform(thrust::device, handle->X.begin(CVD) + size, handle->X.end(CVD), Egnew.HGroup.begin(CVD), handle->X.begin(CVD) + size, []__device__(auto & it1, auto & it2) { return it1 / it2 / it2 / it2; });
		//thrust::copy(handle->X.begin(CVD), handle->X.begin(CVD) + size, Egnew.DeltaGroup.data());
		//thrust::copy(handle->X.begin(CVD) + size, handle->X.end(CVD), Egnew.OmegaGroup.data());
		
		Egnew.omegaGroup.Dvec[size - 1] = 0.0;
		Egnew.omegaGroup.Dvec[size - 2] = -(Egnew.OmegaGroup.Dvec[size - 1] + Egnew.OmegaGroup.Dvec[size - 2]) * Egnew.deltaSGroup.Dvec[size - 1] / 2.0;
		Egnew.velocityGroup.Dvec[size - 1] = 0.0;
		Egnew.velocityGroup.Dvec[size - 2] = (Egnew.DeltaGroup.Dvec[size - 1] + Egnew.DeltaGroup.Dvec[size - 2]) * Egnew.deltaSGroup.Dvec[size - 1] / 2.0;
		velocity_omega_iterate_kernel << <(size + threads - 1) / threads, threads >> > (Egnew.omegaGroup.data(CVD),handle->X.data(CVD) + size,Egnew.deltaSGroup.data(CVD),size);
		velocity_omega_iterate_kernel << <(size + threads - 1) / threads, threads >> > (Egnew.velocityGroup.data(CVD),handle->X.data(CVD), Egnew.deltaSGroup.data(CVD),size);
		hipDeviceSynchronize();
		//0~size-3
		
		thrust::transform_inclusive_scan(Egnew.omegaGroup.rbegin(CVD)+2, Egnew.omegaGroup.rend(CVD), Egnew.omegaGroup.rbegin(CVD)+2, thrust::negate<double>(), thrust::plus<double>());
		thrust::inclusive_scan(Egnew.velocityGroup.rbegin(CVD) + 2, Egnew.velocityGroup.rend(CVD), Egnew.velocityGroup.rbegin(CVD) + 2, thrust::plus<double>());
		velocity_omega_aux_kernel << < (size + threads - 1) / threads, threads >> > (Egnew.velocityGroup.data(CVD), handle->X.data(CVD), Egnew.deltaSGroup.data(CVD), size, thrust::plus<double>());
		velocity_omega_aux_kernel << < (size + threads - 1) / threads, threads >> > (Egnew.omegaGroup.data(CVD), handle->X.data(CVD) + size, Egnew.deltaSGroup.data(CVD), size, thrust::minus<double>());
		hipDeviceSynchronize();
		
		double C;
		//if(model.omegaStandard.first < size - 2)
		C = model.omegaStandard.second - Egnew.omegaGroup.Dvec[model.omegaStandard.first];
		//else 
		//	C = model.omegaStandard.second - Egnew.omegaGroup[model.omegaStandard.first];
		thrust::for_each(Egnew.omegaGroup.begin(CVD), Egnew.omegaGroup.end(CVD), [=]__device__(double& it) {it += C; });
		//Egnew.omegaGroup[size - 1] += C; Egnew.omegaGroup[size - 2] += C;
		//if(model.velocityStandard.first < size - 2)
		C = model.velocityStandard.second - Egnew.velocityGroup.Dvec[model.velocityStandard.first];
		//else
		//	C = model.velocityStandard.second - Egnew.velocityGroup[model.velocityStandard.first];
		thrust::for_each(Egnew.velocityGroup.begin(CVD), Egnew.velocityGroup.end(CVD), [=]__device__(double& it) {it += C; });
		//Egnew.velocityGroup[size - 1] += C; Egnew.velocityGroup[size - 2] += C;
		
		//Egnew.velocityGroup.trans(Egnew.velocityGroup.begin(CVD), Egnew.velocityGroup.end(CVD) - 2, Egnew.velocityGroup.begin());
		//Egnew.omegaGroup.trans(Egnew.omegaGroup.begin(CVD), Egnew.omegaGroup.end(CVD) - 2, Egnew.omegaGroup.begin());
		
	}

}